// compile:
//  v=$(nvidia-smi --query-gpu=compute_cap --format=csv | tail -n1 | tr -cd
//  '[[:digit:]]') nvcc -arch compute_$v -code sm_$v sha.cu -o sha

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define SHR(a, b) (__funnelshift_r((a), 0, (b)))
#define ROT(a, b) (__funnelshift_r((a), (a), (b)))
#define CH(x, y, z) (((x) & ((y) ^ (z))) ^ (z))
#define MAJ(x, y, z) (((x) & ((y) | (z))) | ((y) & (z)))
#define EP0(x) (ROT((x), 2) ^ ROT((x), 13) ^ ROT((x), 22))
#define EP1(x) (ROT((x), 6) ^ ROT((x), 11) ^ ROT((x), 25))
#define SIG0(x) (ROT((x), 7) ^ ROT((x), 18) ^ SHR((x), 3))
#define SIG1(x) (ROT((x), 17) ^ ROT((x), 19) ^ SHR((x), 10))

#define K                                                                      \
  {0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5, 0x3956C25B, 0x59F111F1,     \
   0x923F82A4, 0xAB1C5ED5, 0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3,     \
   0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174, 0xE49B69C1, 0xEFBE4786,     \
   0x0FC19DC6, 0x240CA1CC, 0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,     \
   0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7, 0xC6E00BF3, 0xD5A79147,     \
   0x06CA6351, 0x14292967, 0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13,     \
   0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85, 0xA2BFE8A1, 0xA81A664B,     \
   0xC24B8B70, 0xC76C51A3, 0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,     \
   0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5, 0x391C0CB3, 0x4ED8AA4A,     \
   0x5B9CCA4F, 0x682E6FF3, 0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208,     \
   0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2}

#define B64(x) ((x) + ((x) > 11 ? 53 + ((x) > 37) * 6 : (46 - (!(x)) * 3)))

#define W(a, b, c, d) d, c, b, a
#define MSG                                                                    \
  W('s', 'e', 'l', 'e'), W('t', 's', 'k', 'i'), W('y', '/', 'H', 'i'),         \
      W('r', 'e', 'M', 'e'), W('/', 'H', 'i', 'H'), W('N', '/', 'H', 'i'),     \
      W('A', 'l', 'e', 'x'), W('/', '2', '5', 'G'), W('H', 's', '/', 'R'),     \
      W('T', 'X', '4', '0'), W('9', '0', '/', '0'), 0, '_', '_', '_', 0, '_',  \
      '_', '_', 0x80, 0, 0, 0, 0x00, 0x00, 0x00, 0x00, 0xb8, 0x01, 0x00, 0x00, \
                                                                               \
      0x3c, 0x77, 0x7a, 0xce, 0x6f, 0x84, 0x5c, 0xbe

__global__ void kernel(uint32_t _a, uint32_t _b, uint32_t _c, uint32_t _d,
                       uint32_t _e, uint32_t _f, uint32_t _g, uint32_t _h,
                       uint32_t _m_18, uint32_t batch, uint64_t chunk,
                       uint8_t *block, uint32_t *target, uint32_t *mutex) {
  uint8_t msg[32 * 16] = {MSG};

  static const __constant__ uint32_t k[64] = K;

  *((uint32_t *)&msg[44]) = batch;
  msg[48] = blockIdx.x;
  msg[49] = blockIdx.y;
  msg[50] = blockIdx.z;
  msg[51] = threadIdx.x;
  msg[53] = threadIdx.y;
  *((uint64_t *)&msg[48]) += chunk;

  uint64_t target_a = ~0;

  for (uint8_t u = 0x41; u < 0x59; u++) {
    for (uint8_t v = 0x41; v < 0x59; v++) {
      msg[54] = u;
      msg[55] = v;

      uint32_t a, b, c, d, e, f, g, h, t1, t2;

      a = _a;
      b = _b;
      c = _c;
      d = _d;
      e = _e;
      f = _f;
      g = _g;
      h = _h;

      uint32_t i = 0, *m = (uint32_t *)msg;

#pragma unroll 4
      for (i = 12; i < 16; ++i) {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
        //if (i == 10) { printf("a = 0x%08x;\n", a); printf("b = 0x%08x;\n", b); printf("c = 0x%08x;\n", c); \
        printf("d = 0x%08x;\n", d); printf("e = 0x%08x;\n", e); printf("f = 0x%08x;\n", f); \
        printf("g = 0x%08x;\n", g); printf("h = 0x%08x;\n", h); }
      }

      /* m[16] = _m_16; */
      /* m[17] = _m_17; */
      /* m[16] = 0xce7a773c; */
      /* m[17] = 0xbe5c846f; */
      m[18] = _m_18;

#pragma unroll 43
      for (i = 19; i < 62; ++i) {
        m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
        /* if (i==18) printf("%08x\n", m[i]); */
        // if (i < 18) { printf("m[%d] = 0x%08x;\n", i, m[i]); }
      }

#pragma unroll 44
      for (i = 16; i < 60; ++i) {
        t1 = h + EP1(e) + CH(e, f, g) + k[i] + m[i];
        t2 = EP0(a) + MAJ(a, b, c);
        h = g;
        g = f;
        f = e;
        e = d + t1;
        d = c;
        c = b;
        b = a;
        a = t1 + t2;
      }

      // unroll four iterations to avoid writing to m[i] and other extra
      // variables
      t1 = h + EP1(e) + CH(e, f, g) + k[60] + m[60];

      h = d + t1;
      d = t1 + EP0(a) + MAJ(a, b, c);

      g = g + EP1(h) + CH(h, e, f) + k[61] + m[61];
      c = c + g;

      t1 = f + EP1(c) + CH(c, h, e) + k[62] + SIG1(m[60]) + m[55] +
           SIG0(m[47]) + m[46];
      t2 = EP0(d) + MAJ(d, a, b) + g;

      f = t1 + EP0(t2) + MAJ(t2, d, a);
      // we are interested in checking second part of the hash first
      if (uint64_t(f + 0xbb67ae85) < *target) {
        t1 = t1 + b;
        b = e + EP1(t1) + CH(t1, c, h) + k[63] + SIG1(m[61]) + m[56] +
            SIG0(m[48]) + m[47] + EP0(f) + MAJ(f, t2, d);

        if (!uint64_t(b - 0x95f61999)) {
          target_a =
              (uint64_t(f + 0xbb67ae85)) + (uint64_t(b - 0x95f61999) << 32);
          goto found;
        }
      }
      // if (target_a < *target) goto found;
    }
  }
found:

  if (target_a > *target)
    return;

  while (atomicCAS(mutex, 0, 1) != 0)
    ;
  if (target_a < *target) {
    *target = target_a;
    block[3] = msg[44];
    block[2] = msg[45];
    block[1] = msg[46];
    block[0] = msg[47];

    block[7] = msg[48];
    block[6] = msg[49];
    block[5] = msg[50];
    block[4] = msg[51];

    block[11] = 0;
    block[10] = msg[53];
    block[9] = msg[54];
    block[8] = msg[55];
  }
  atomicExch(mutex, 0);
}

#define cudaNoError(expr)                                                      \
  { cudaAssert((expr), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "cudaAssert: %s %s:%d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

int main(int argc, char *argv[]) {
  setvbuf(stdout, NULL, _IONBF, 0);

  uint32_t shard;
  uint32_t target, batch;

  if (argc < 3)
    fprintf(stderr, "usage: %s <shard> <target> <batch start>\n", argv[0]),
        exit(1);
  if (sscanf(argv[1], "%x", &shard) == EOF)
    exit(2);
  if (sscanf(argv[2], "%x", &target) == EOF)
    exit(2);
  if (sscanf(argv[3], "%x", &batch) == EOF)
    exit(2);

  fprintf(stderr, "* shard %02x | target start %08x | batch start %06x\n",
          shard, target, batch);

  uint8_t block[12] = {0}, *block_cu;

  const uint8_t chunks_n = 3;
  uint32_t *target_cu[chunks_n], *mutex_cu[chunks_n];

  cudaNoError(hipMalloc(&block_cu, sizeof(uint8_t) * 12));

  for (uint8_t j = 0; j < chunks_n; j++) {
    cudaNoError(hipMalloc(&target_cu[j], sizeof(uint32_t)));
    cudaNoError(hipMalloc(&mutex_cu[j], sizeof(uint32_t)));
  }

  for (uint8_t j = 0; j < chunks_n; j++) {
    cudaNoError(hipMemcpy(target_cu[j], &target, sizeof(uint32_t),
                           hipMemcpyHostToDevice));
    cudaNoError(hipMemset(mutex_cu[j], 0, sizeof(uint32_t)));
  }

  cudaNoError(hipMemset(block_cu, 0, sizeof(uint8_t) * 12));

  const uint64_t batches_n = 64 * 64 * 64;

  static const uint32_t k[64] = K;

  uint32_t a_0 = 0x6a09e667;
  uint32_t b_0 = 0xbb67ae85;
  uint32_t c_0 = 0x3c6ef372;
  uint32_t d_0 = 0xa54ff53a;
  uint32_t e_0 = 0x510e527f;
  uint32_t f_0 = 0x9b05688c;
  uint32_t g_0 = 0x1f83d9ab;
  uint32_t h_0 = 0x5be0cd19;

  /* #undef W */
  /* #define W(a, b, c, d) a, b, c, d */
  uint8_t msg[64 + 4 * 4] = {MSG};
  /* #undef W */

  for (; batch < batches_n; batch++) {
    struct timespec t_start, t_end;
    uint64_t batch_size = 0;

    msg[44] = B64((batch >> 0) & 0x3f);
    msg[45] = B64((batch >> 6) & 0x3f);
    msg[46] = B64((batch >> 12) & 0x3f);
    msg[47] = B64(shard);

#undef ROT
#undef SHR
#define ROT(a, b) (((a) >> (b)) | ((a) << (32 - (b))))
#define SHR(a, b) ((a) >> (b))

    uint32_t a = a_0;
    uint32_t b = b_0;
    uint32_t c = c_0;
    uint32_t d = d_0;
    uint32_t e = e_0;
    uint32_t f = f_0;
    uint32_t g = g_0;
    uint32_t h = h_0;
    for (uint8_t i = 0; i < 12; ++i) {
      uint32_t t1 = h + EP1(e) + CH(e, f, g) + k[i] + ((uint32_t *)(msg))[i];
      uint32_t t2 = EP0(a) + MAJ(a, b, c);
      h = g;
      g = f;
      f = e;
      e = d + t1;
      d = c;
      c = b;
      b = a;
      a = t1 + t2;
    }

    uint32_t *m = (uint32_t *)msg;
    /* m[16] = 0xce7a773c; */
    /* m[17] = 0xbe5c846f; */
    for (uint32_t i = 16; i < 19; i++)
      m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

#undef ROT
#undef SHR

    clock_gettime(CLOCK_MONOTONIC, &t_start);
    {
      uint64_t chunks[3] = {
          0x4141410041414141,
          0x6161610061616161,
          0x2f2f2f002f2f2f2f,
      };

      for (uint8_t x = 0; x < chunks_n; x++) {
        uint8_t len = x == 2 ? 11 : 26;
        dim3 blocks(len, len, len);
        dim3 threads(len, len, 1);

        batch_size += len * len * len * len * len;

        uint64_t chunk = chunks[x];
        kernel<<<blocks, threads>>>(a, b, c, d, e, f, g, h, m[18],
                                    *(uint32_t *)(&msg[44]), chunk, block_cu,
                                    target_cu[x], mutex_cu[x]);

        cudaNoError(hipPeekAtLastError());
      }

      for (uint8_t x = 0; x < chunks_n; x++) {
        uint32_t new_target;
        cudaNoError(hipMemcpy(&new_target, target_cu[x], sizeof(uint32_t),
                               hipMemcpyDeviceToHost));
        if (new_target < target) {
          target = new_target;
          cudaNoError(hipMemcpy(block, block_cu, sizeof(uint8_t) * 12,
                                 hipMemcpyDeviceToHost));
          printf("%08x ", target);
          for (uint8_t u = 0; u < 44; u += 4)
            printf("%c%c%c%c", msg[u + 3], msg[u + 2], msg[u + 1], msg[u + 0]);
          printf("%s\n", block);
        }
      }
    }

    clock_gettime(CLOCK_MONOTONIC, &t_end);
    double time_taken = (t_end.tv_sec - t_start.tv_sec) +
                        (t_end.tv_nsec - t_start.tv_nsec) * 1e-9;
    fprintf(stderr, "* %06x [%s] %3.2fs %3.2fGk/s | %08x => %s\n", batch,
            &msg[44], time_taken,
            batch_size / time_taken / 1000 / 1000 / 1000 * 26 * 26, target,
            block[0] > 0 ? (const char *)block : "<none>");
  }

  cudaNoError(hipFree(block_cu));
  cudaNoError(hipFree(target_cu));
  cudaNoError(hipFree(mutex_cu));

  return 0;
}
